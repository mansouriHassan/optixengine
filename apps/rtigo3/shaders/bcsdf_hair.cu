#include "hip/hip_runtime.h"

/* ---------------------------------------------------------
* Bidirectional curve scattering distribution function
* for a hair material
* ---------------------------------------------------------
* Based on the hair rendering implementation from the
* tungsten renderer for "energy-conserving hair reflectance
* model" and importance sampling for physically-based hair
* fiber models" from dEon et al.
* ---------------------------------------------------------
*/

/* ----------------------- Libraries ----------------------- */

#include <stdio.h>
#include "config.h"

#include <optix.h>

#include "per_ray_data.h"
#include "material_definition.h"
#include "shader_common.h"
#include "random_number_generators.h"
#include "curve.h"

/*rtBuffer<float3> id_values_sop;
rtBuffer<float3> id_values_cop;*/

/* ----------------------- Functions ----------------------- */

__forceinline__ __device__ float logI0(float const& x) {
	return (x > 12.0f) ? x + 0.5f * (-logf((2.f * M_PIf * x)) + fdividef(1.0f, (8.0f * x))) : logf(cyl_bessel_i0f(x));
}

__forceinline__ __device__ float FrDielectric(float const& cos_theta, float const& n1,
	float const& n2) {

	float const R0 = fdividef(n1 - n2, n1 + n2) * fdividef(n1 - n2, n1 + n2);
	return R0 + (1.f - R0) * (1.f - cos_theta) * (1.f - cos_theta) * (1.f - cos_theta) * (1.f - cos_theta) * (1.f - cos_theta);
}


/* Rough longitudinal scattering function with variance
* v = beta^2 ---------------------------------------------- */
__forceinline__ __device__ float M(float const& v, float const& sin_theta_i,
	float const& sin_theta_o, float const& cos_theta_i,
	float const& cos_theta_o) {
	float a = cos_theta_i * fdividef(cos_theta_o, v);
	float b = sin_theta_i * fdividef(sin_theta_o, v);
	float mp =
		(v <= 0.1f)
		? (expf(logI0(a) - b - fdividef(1.f, v) + 0.6931f + logf(fdividef(1.f, (2.f * v)))))
		: fdividef((expf(-b) * cyl_bessel_i0f(a)), (sinhf(fdividef(1.f, v)) * 2.f * v));
	return mp;
}


/* Sampling methods ----------------------------------------- */
__forceinline__ __device__ float3 ApR(float const& h, float const& cos_theta_o,
	float const& eta) {

	float cosgamma0 = sqrtf(1.f - h * h);
	float cosTheta = cos_theta_o * cosgamma0;
	float f = FrDielectric(cosTheta, 1.f, eta);
	return make_float3(f);
}

__forceinline__ __device__ float Phi(int const& p, float const& gammat,
	float const& gamma0) {

	return 2.f * p * gammat - 2.f * gamma0 + p * M_PIf;
}

__forceinline__ __device__ float Logistic(float const& x,
	float const& s) {
	float ax = fabsf(x);
	float frac = fdividef(ax, s);
	return expf(-frac) / (s * (1.f + expf(-frac)) * (1.f + expf(-frac)));
}

__forceinline__ __device__ float LogisticCDF(float const& x, float const& s) {
	return fdividef(1.f, (1.f + expf(-fdividef(x, s))));
}

__forceinline__ __device__ float TrimmedLogistic(float const& x, float const& s, float const& a, float const& b) {
	return fdividef(Logistic(x, s), (LogisticCDF(b, s) - LogisticCDF(a, s)));
}

__forceinline__ __device__ float SampleTrimmedLogistic(float const& u, float const& s, float const& a, float const& b) {
	float k = LogisticCDF(b, s) - LogisticCDF(a, s);
	float x = -s * logf(fdividef(1.f, (u * k + LogisticCDF(a, s))) - 1.f);
	return clamp(x, a, b);
}

__forceinline__ __device__ float Np(float const& phi,
	int const& p, float const& s, float const& gammaO, float const& gammaT) {
	float dphi = phi - Phi(p, gammaT, gammaO);
	// Remap _dphi_ to $[-\pi,\pi]$
	while (dphi > M_PIf) { dphi -= 2.f * M_PIf; }
	while (dphi < -M_PIf) { dphi += 2.f * M_PIf; }
	return TrimmedLogistic(dphi, s, -M_PIf, M_PIf);
}

extern "C" __device__ void __direct_callable__sample_bcsdf_hair(MaterialDefinition const& material, State const& state, PerRayData * prd) {
	// Conventional wi correspond to -wo in the code
	// Conventional wo correspond to wi in the code

	
	float h = -1.f + 2.f * rng(prd->seed);//dot(state.normal, cross(state.tangent, state.texcoord)); //
	float gammaO = asinf(h);

	float2 xi_N = rng2(prd->seed);
	float2 xi_M = rng2(prd->seed);

	float ior = 1.55f;

	const float sin_theta_o = dot(prd->wo, state.tangent);
	const float cos_theta_o = trigInverse(sin_theta_o);
	const float ndwo = dot(prd->wo, state.texcoord);
	const float bdwo = dot(prd->wo, normalize(cross(state.texcoord,state.tangent)));
	float phio = atan2f(prd->wo.x, prd->wo.z);

	float sin2kAlpha0 = sinf(material.scale_angle_rad);
    float cos2kAlpha0 = trigInverse(sin2kAlpha0);
	float sin2kAlpha1 = 2.f * cos2kAlpha0 * sin2kAlpha0;
    float cos2kAlpha1 = cos2kAlpha0*cos2kAlpha0-sin2kAlpha0*sin2kAlpha0;
	float sin2kAlpha2 = 2.f * cos2kAlpha1 * sin2kAlpha1;
	float cos2kAlpha2 = cos2kAlpha1 * cos2kAlpha1 - sin2kAlpha1 * sin2kAlpha1;



	float sinThetaT = sin_theta_o / ior;
	float cosThetaT = trigInverse(sinThetaT);
	float etap = sqrtf(ior * ior - (sin_theta_o * sin_theta_o)) / cos_theta_o;
	//float etas = ior * ior*cos_theta_o/sqrtf(ior * ior - (sin_theta_o*sin_theta_o));
	float sinGammaT = h / etap;
	float cosGammaT = trigInverse(sinGammaT);
	float gammaT = asinf(sinGammaT);


	float3 absorption = material.absorption;
	const float3 absorption_eumelanin = make_float3(0.419f, 0.697f, 1.37f);
	const float3 absorption_pheomelanin = make_float3(0.187f, 0.4f, 1.05f);

	float melanin_ratio = clamp(material.melanin_ratio * (1.f + material.melanin_ratio_disparity * state.rand.y),0.f,1.f);
	float melanin_concentration = fmaxf(material.melanin_concentration * (1.f + state.rand.y * material.melanin_concentration_disparity),.0f);
	absorption += (state.rand.x > material.whitepercen) ? melanin_concentration * lerp(absorption_eumelanin, absorption_pheomelanin, melanin_ratio) : make_float3(0.f);

	float3 T = expf(-absorption * 2.f * cosGammaT / cosThetaT);

	float3 R = ApR(h, cos_theta_o, etap);
	//float3 Rs = ApR(h, cos_theta_o,etas);
	//float3 R = 0.5f*(Rp+Rs);
	float3 TT = (make_float3(1.f) - R) * (make_float3(1.f) - R) * T;
	float3 TRT = TT * R * T;
	//float3 TRRT = TRT*T*R/(1.f-T*R);

	float lum = luminance(R) + luminance(TT) + luminance(TRT);
	float3 apsample = make_float3(luminance(R), luminance(TT), luminance(TRT)) / lum;

	//Pour ajouter un 4e lobe

	//float lum =  luminance(R)  +luminance(TT) + luminance(TRT)+luminance(TRRT) ;
	//float4 apsample =  make_float4(luminance(R),luminance(TT),luminance(TRT),luminance(TRRT))/lum;

	const float s = 0.626657069f * (0.265f * material.betaN + 1.194f * (material.betaN * material.betaN) + 5.372f * (material.betaN * material.betaN) * (material.betaN * material.betaN) * (material.betaN * material.betaN) *
		(material.betaN * material.betaN) * (material.betaN * material.betaN) * (material.betaN * material.betaN) * (material.betaN * material.betaN) * (material.betaN * material.betaN) *
		(material.betaN * material.betaN) * (material.betaN * material.betaN) * (material.betaN * material.betaN));

	float dphi = SampleTrimmedLogistic(xi_N.y, s, -M_PIf, M_PIf);

	float cosPhi = cosf(2.f * M_PIf * xi_M.y);

	//Sample M
	const float sqrtv = 0.726f * material.betaM + 0.812f * (material.betaM * material.betaM) + 3.7f * (material.betaM * material.betaM) * (material.betaM * material.betaM) * (material.betaM * material.betaM) *
		(material.betaM * material.betaM) * (material.betaM * material.betaM) * (material.betaM * material.betaM) * (material.betaM * material.betaM) * (material.betaM * material.betaM) *
		(material.betaM * material.betaM) * (material.betaM * material.betaM);

	dphi += (xi_N.x < apsample.x) ? Phi(0, gammaO, gammaT) : (xi_N.x < apsample.x + apsample.y) ? Phi(1, gammaO, gammaT) : Phi(2, gammaO, gammaT);
	float phiI = phio + dphi;

	//mod�le non s�parable

	float theta_0 = sin_theta_o - 2.f * sin2kAlpha0 * (cosf(phio * 0.5f) * cos2kAlpha0 * cos_theta_o + sin_theta_o * sin2kAlpha0);
	float sinThetaOp0 = sinf(theta_0);// sin_theta_o*cos2kAlpha1-cos_theta_o*sin2kAlpha1;
	float cosThetaOp0 = cosf(theta_0);//  cos_theta_o*cos2kAlpha1+sin_theta_o*sin2kAlpha1;

	//Mod�le s�parable
	//float sinThetaOp0 = sin_theta_o*cos2kAlpha1-cos_theta_o*sin2kAlpha1;
	//float cosThetaOp0 = cos_theta_o*cos2kAlpha1+sin_theta_o*sin2kAlpha1;


	float sinThetaOp1 = sin_theta_o * cos2kAlpha0 + cos_theta_o * sin2kAlpha0;
	float cosThetaOp1 = cos_theta_o * cos2kAlpha0 - sin_theta_o * sin2kAlpha0;
	float sinThetaOp2 = sin_theta_o * cos2kAlpha2 + cos_theta_o * sin2kAlpha2;
	float cosThetaOp2 = cos_theta_o * cos2kAlpha2 - sin_theta_o * sin2kAlpha2;

	float coef = (xi_N.x < apsample.x) ? 1.0f : (xi_N.x < apsample.x + apsample.y) ? 0.25f : 4.f;

	float cosTheta = 1.f + coef * sqrtv * sqrtv * logf(xi_M.x + (1.f - xi_M.x) * expf(-2.f / (coef * sqrtv * sqrtv)));

	float sinTheta = trigInverse(cosTheta);



	float sinThetaI = (xi_N.x < apsample.x) ? -cosTheta * sinThetaOp0 + sinTheta * cosPhi * cosThetaOp0
		: (xi_N.x < apsample.x + apsample.y) ? -cosTheta * sinThetaOp1 + sinTheta * cosPhi * cosThetaOp1
		: -cosTheta * sinThetaOp2 + sinTheta * cosPhi * cosThetaOp2;
	float cosThetaI = trigInverse(sinThetaI);


	//prd->wi = normalize(make_float3( sinThetaI, cosThetaI * cosf(phiI), cosThetaI * sinf(phiI)));
	prd->wi = normalize(make_float3(cosThetaI * sinf(phiI), sinThetaI, cosThetaI * cosf(phiI)));
	// Evaluate longitudinal scattering functions
	cosThetaOp0 = fabsf(cosThetaOp0);
	cosThetaOp1 = fabsf(cosThetaOp1);
	cosThetaOp2 = fabsf(cosThetaOp2);

	const float M_R = M(sqrtv * sqrtv, sinThetaI, sinThetaOp0, cosThetaI, cosThetaOp0);
	const float M_TT = M(0.25f * sqrtv * sqrtv, sinThetaI, sinThetaOp1, cosThetaI, cosThetaOp1);
	const float M_TRT = M(sqrtv * sqrtv * 4.f, sinThetaI, sinThetaOp2, cosThetaI, cosThetaOp2);
	//const float M_TRRT = M(sqrtv*sqrtv*4.f, sinThetaI, fabsf(sin_theta_o),cosThetaI, fabsf(cos_theta_o));

	const float N_R = Np(dphi, 0, s, gammaO, gammaT);//0.25f*abs(cosf(0.5f*(dphi)));//
	const float N_TT = Np(dphi, 1, s, gammaO, gammaT);
	const float N_TRT = Np(dphi, 2, s, gammaO, gammaT);


	prd->pdf = M_R* N_R* apsample.x + M_TT * N_TT * apsample.y + M_TRT * N_TRT * apsample.z;//+M_TRRT*apsample.w*0.5f*M_1_PIf;

	prd->wi = prd->wi.x* state.texcoord + prd->wi.y * state.tangent + prd->wi.z * normalize(cross(state.texcoord, state.tangent));

	
	if (dot(prd->wi, state.normal) < 0.f)
	{
		prd->pos = prd->pos - 2.f * state.normal * state.radius;
	}
	
	prd->f_over_pdf = (M_R * N_R * R + M_TT * N_TT * TT + N_TRT * M_TRT * TRT/* +M_TRRT*TRRT*0.5*M_1_PIf*/) / prd->pdf;

	prd->flags |= FLAG_DIFFUSE;
	
}


/* Azimuthal and logitudinal evaluation for given theta and phi */
extern "C" __device__ float4 __direct_callable__eval_bcsdf_hair(MaterialDefinition const& material, State const& state, PerRayData* const prd, const float3 wiL) {

	float ior = 1.55f;

	float h = -1.f + 2.f * rng(prd->seed);
	float gammaO = asinf(h);


	const float sin_theta_o = dot(prd->wo, state.tangent);
	const float cos_theta_o = trigInverse(sin_theta_o);
	const float ndwo = dot(prd->wo, state.texcoord);
	const float bdwo = dot(prd->wo, cross(state.texcoord, state.tangent));
	float phio = atan2f(ndwo, bdwo);//
	//float phio = asinf(dot(prd->wo, state.normal));

	const float sin_theta_i = dot(wiL, state.tangent);
	const float cos_theta_i = trigInverse(sin_theta_i);
	const float ndwi = dot(wiL, state.texcoord);
	const float bdwi = dot(wiL, cross(state.texcoord, state.tangent));
	float phii = atan2f(ndwi, bdwi);//
	//float phii = asinf(dot(wiL, state.normal));

	// Compute $\cos \thetat$ for refracted ray
	float sinThetaT = sin_theta_o / ior;
	float cosThetaT = trigInverse(sinThetaT);

	// Compute $\gammat$ for refracted ray
	float etap = sqrtf(ior * ior - (sin_theta_o * sin_theta_o)) / cos_theta_o;
	float sinGammaT = h / etap;
	float cosGammaT = trigInverse(sinGammaT);
	float gammaT = asinf(sinGammaT);

	float phi = phii - phio;

	float3 absorption = material.absorption;
	const float3 absorption_eumelanin = make_float3(0.419f, 0.697f, 1.37f);
	const float3 absorption_pheomelanin = make_float3(0.187f, 0.4f, 1.05f);

	float melanin_ratio = clamp(material.melanin_ratio * (1.f + material.melanin_ratio_disparity * state.rand.y), 0.f, 1.f);
	float melanin_concentration = fmaxf(material.melanin_concentration * (1.f + state.rand.y * material.melanin_concentration_disparity), .0f);
	absorption += (state.rand.x > material.whitepercen) ? melanin_concentration * lerp(absorption_eumelanin, absorption_pheomelanin, melanin_ratio) : make_float3(0.f);


	float3 T = expf(-absorption * 2.f * cosGammaT / cosThetaT);
	float3 R = ApR(h, cos_theta_o, etap);
	//float3 Rs = ApR(h, cos_theta_o,etas);
	//float3 R = 0.5f*(Rp+Rs);
	float3 TT = (make_float3(1.f) - R) * (make_float3(1.f) - R) * T;
	float3 TRT = TT * R * T;
	//float3 TRRT = TRT*T*R/(1.f-T*R);

	//float lum =  luminance(R)  +luminance(TT) + luminance(TRT)+luminance(TRRT) ;
	//float4 apsample =  make_float4(luminance(R),luminance(TT),luminance(TRT),luminance(TRRT))/lum;

	float lum = luminance(R) + luminance(TT) + luminance(TRT);
	float3 apsample = make_float3(luminance(R), luminance(TT), luminance(TRT)) / lum;

	float sin2kAlpha0 = sinf(material.scale_angle_rad);
	float cos2kAlpha0 = trigInverse(sin2kAlpha0);
	float sin2kAlpha1 = 2.f * cos2kAlpha0 * sin2kAlpha0;
	float cos2kAlpha1 = cos2kAlpha0 * cos2kAlpha0 - sin2kAlpha0 * sin2kAlpha0;
	float sin2kAlpha2 = 2.f * cos2kAlpha1 * sin2kAlpha1;
	float cos2kAlpha2 = cos2kAlpha1 * cos2kAlpha1 - sin2kAlpha1 * sin2kAlpha1;

	float theta_0 = sin_theta_o - 2.f * sin2kAlpha0 * (cosf(phio * 0.5f) * cos2kAlpha0 * cos_theta_o + sin_theta_o * sin2kAlpha0);
	float sinThetaOp0 = sinf(theta_0);// sin_theta_o*cos2kAlpha1-cos_theta_o*sin2kAlpha1;
	float cosThetaOp0 = cosf(theta_0);//  cos_theta_o*cos2kAlpha1+sin_theta_o*sin2kAlpha1;

	//float sinThetaOp0 = sin_theta_o*cos2kAlpha1-cos_theta_o*sin2kAlpha1;
	//float cosThetaOp0 = cos_theta_o*cos2kAlpha1+sin_theta_o*sin2kAlpha1;

	float sinThetaOp1 = sin_theta_o * cos2kAlpha0 + cos_theta_o * sin2kAlpha0;
	float cosThetaOp1 = cos_theta_o * cos2kAlpha0 - sin_theta_o * sin2kAlpha0;

	float sinThetaOp2 = sin_theta_o * cos2kAlpha2 + cos_theta_o * sin2kAlpha2;
	float cosThetaOp2 = cos_theta_o * cos2kAlpha2 - sin_theta_o * sin2kAlpha2;


	const float sqrtv = 0.726f * material.betaM + 0.812f * (material.betaM * material.betaM) + 3.7f * (material.betaM * material.betaM) * (material.betaM * material.betaM) * (material.betaM * material.betaM) *
		(material.betaM * material.betaM) * (material.betaM * material.betaM) * (material.betaM * material.betaM) * (material.betaM * material.betaM) * (material.betaM * material.betaM) *
		(material.betaM * material.betaM) * (material.betaM * material.betaM);

	const float s = 0.626657069f * (0.265f * material.betaN + 1.194f * (material.betaN * material.betaN) + 5.372f * (material.betaN * material.betaN) * (material.betaN * material.betaN) * (material.betaN * material.betaN) *
		(material.betaN * material.betaN) * (material.betaN * material.betaN) * (material.betaN * material.betaN) * (material.betaN * material.betaN) * (material.betaN * material.betaN) *
		(material.betaN * material.betaN) * (material.betaN * material.betaN) * (material.betaN * material.betaN));

	// Evaluate longitudinal scattering functions
	cosThetaOp0 = fabsf(cosThetaOp0);
	cosThetaOp1 = fabsf(cosThetaOp1);
	cosThetaOp2 = fabsf(cosThetaOp2);

	const float M_R = M(sqrtv * sqrtv, sin_theta_i, sinThetaOp0, cos_theta_i, cosThetaOp0);
	const float M_TT = M(0.25f * sqrtv * sqrtv, sin_theta_i, sinThetaOp1, cos_theta_i, cosThetaOp1);
	const float M_TRT = M(sqrtv * sqrtv * 4.f, sin_theta_i, sinThetaOp2, cos_theta_i, cosThetaOp2);
	//const float M_TRRT = M(sqrtv*sqrtv*4.f, sin_theta_i, sin_theta_o,cos_theta_i, cos_theta_o);

	const float N_R = Np(phi, 0.f, s, gammaO, gammaT);
	const float N_TT = Np(phi, 1.f, s, gammaO, gammaT);
	const float N_TRT = Np(phi, 2.f, s, gammaO, gammaT);

	const float3 f = M_R * N_R * R + M_TT * N_TT * TT + N_TRT * M_TRT * TRT;//+M_TRRT*TRRT*0.5*M_1_PIf;
	float pdf = M_R * N_R * apsample.x + M_TT * N_TT * apsample.y + M_TRT * N_TRT * apsample.z;//+M_TRRT*apsample.w*0.5f*M_1_PIf;
	return make_float4(f, pdf);
}